#include "hip/hip_runtime.h"
//===------------------------------------------------------------*- C++ -*-===//
//
//             Ripples: A C++ Library for Influence Maximization
//                  Marco Minutoli <marco.minutoli@pnnl.gov>
//                   Pacific Northwest National Laboratory
//
//===----------------------------------------------------------------------===//
//
// Copyright (c) 2019, Battelle Memorial Institute
//
// Battelle Memorial Institute (hereinafter Battelle) hereby grants permission
// to any person or entity lawfully obtaining a copy of this software and
// associated documentation files (hereinafter “the Software”) to redistribute
// and use the Software in source and binary forms, with or without
// modification.  Such person or entity may use, copy, modify, merge, publish,
// distribute, sublicense, and/or sell copies of the Software, and may permit
// others to do so, subject to the following conditions:
//
// 1. Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimers.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Other than as used herein, neither the name Battelle Memorial Institute or
//    Battelle may be used in any form whatsoever without the express written
//    consent of Battelle.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL BATTELLE OR CONTRIBUTORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
//===----------------------------------------------------------------------===//

#include "ripples/cuda/cuda_graph.cuh"
#include "ripples/cuda/cuda_hc_engine.h"
#include "ripples/cuda/cuda_supported_graphs.h"

#include "trng/lcg64.hpp"
#include "trng/uniform01_dist.hpp"

namespace ripples {
__inline__ __device__
int warpReduceSum(int val) {
  #define FULL_MASK 0xffffffff

  for (int offset = warpSize/2; offset > 0; offset /= 2)
    val += __shfl_down_sync(FULL_MASK, val, offset);

  return val;
}

template <typename GraphTy, typename PRNGTy>
__global__ void generate_sample_ic_kernel(
    size_t batch_size, size_t num_edges,
    typename cuda_device_graph<GraphTy>::weight_t *weights,
    PRNGTy *d_trng_states, int *d_flag) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  trng::uniform01_dist<float> u;
  auto &r(d_trng_states[tid]);

  while (batch_size > 0) {
  int limit = (num_edges / (8 * sizeof(int)) + 1) * 8 *sizeof(int);
  for (int pos = tid; pos < limit; pos += blockDim.x * gridDim.x) {
    typename cuda_device_graph<GraphTy>::weight_t w = weights[pos];
    int edge_flag = u(r) <= w ? 1 << (tid % warpSize) : 0;
    edge_flag = warpReduceSum(edge_flag);
    if ((tid % warpSize) == 0)
      d_flag[pos / warpSize] = edge_flag;

  }
  d_flag += (num_edges / (8 * sizeof(int))) + 1;
  --batch_size;
  }
}

template <typename GraphTy, typename PRNGTy>
void cuda_generate_samples_ic(size_t n_blocks, size_t block_size,
                              size_t batch_size, size_t num_edges,
                              PRNGTy *d_trng_states, cuda_ctx<GraphTy> *ctx,
                              int *d_flags, hipStream_t stream) {
  generate_sample_ic_kernel<GraphTy, PRNGTy>
      <<<n_blocks, block_size, 0, stream>>>(batch_size, num_edges,
                                            ctx->d_graph->d_weights_,
                                            d_trng_states, d_flags);
  cuda_check(__FILE__, __LINE__);
}

template <typename GraphTy, typename PRNGTy>
void cuda_generate_samples_lt(size_t n_blocks, size_t block_size,
                              size_t batch_size, size_t num_edges,
                              PRNGTy *d_trng_states, cuda_ctx<GraphTy> *ctx,
                              int *d_flags, hipStream_t stream) {}

template void cuda_generate_samples_lt<HCGraphTy, trng::lcg64>(
    size_t n_blocks, size_t block_size, size_t batch_size, size_t num_edges,
    trng::lcg64 *d_trng_states, cuda_ctx<HCGraphTy> *ctx, int *d_flags,
    hipStream_t stream);
template void cuda_generate_samples_ic<HCGraphTy, trng::lcg64>(
    size_t n_blocks, size_t block_size, size_t batch_size, size_t num_edges,
    trng::lcg64 *d_trng_states, cuda_ctx<HCGraphTy> *ctx, int *d_flags,
    hipStream_t stream);
}  // namespace ripples
